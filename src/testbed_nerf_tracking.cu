#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#include <cmath>

#include <testbed_nerf_utils.cu>

#ifdef copysign
#undef copysign
#endif

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN


// inline __device__ MatrixXf make_gaussian_kernel(const uint32_t kernel_size) {
//
// 	MatrixXf kernel(kernel_size, kernel_size);
//
//     if (kernel_size==1) {
//         kernel(0,0) = 1.f;
//     } else {
//         VectorXf p(kernel_size);
//         switch (kernel_size) {
//             case 3:
//                 p[0]=1; p[1]=2; p[2]=1;
//                 break;
//             case 5:
//                 p[0]=1; p[1]=4; p[2]=6; p[3]=4; p[4]=1;
//                 break;
//             case 7:
//                 p[0]=1; p[1]=6; p[2]=15; p[3]=20; p[4]=15; p[5]=6; p[6]=1;
//                 break;
//             case 9:
//                 p[0]=1; p[1]=8; p[2]=28; p[3]=56; p[4]=70; p[5]=56; p[6]=28; p[7]=8; p[8]=1;
//                 break;
//             case 11:
//                 p[0]=1; p[1]=10; p[2]=45; p[3]=120; p[4]=210; p[5]=252; p[6]=210; p[7]=120; p[8]=45; p[9]=10; p[10]=1;
//                 break;
//         }
//         p = p / p.sum();
//         kernel = p*p.transpose();
//     }
//
//     return kernel;
// }


std::vector<float> Testbed::make_gaussian_kernel_debug(const uint32_t kernel_size, const float sigma) {

    std::vector<float> kernel(kernel_size * kernel_size);

    if (kernel_size==1) {
        kernel[0] = 1.f;
        return kernel;
    }

    uint32_t hw = kernel_size / 2;

    const double pi = 3.14159265358979323846;
    uint32_t cpt=0;
    for (uint32_t i=0; i < kernel_size; i++){
        for (uint32_t j=0; j < kernel_size; j++){

            float g = 1/(2*pi*sigma*sigma) * std::exp( -( static_cast<float>( (i-hw)*(i-hw) + (j-hw)*(j-hw) ) ) / static_cast<float>(2*sigma*sigma) );

            kernel[cpt] = g;
            ++cpt;
        }
    }
    return kernel;
}

__global__ void sample_training_pixels_for_tracking(
	const uint32_t n_rays,
    const uint32_t ray_stride,
	default_rng_t rng,
    const uint32_t kernel_window_size,
    const uint32_t sample_away_from_border_margin_h,
    const uint32_t sample_away_from_border_margin_w,
	const TrainingImageMetadata* __restrict__ metadata,
	const uint32_t indice_image_for_tracking_pose,
	bool snap_to_pixel_centers,
    float* __restrict__ xy_image_pixel_indices
) {
	const uint32_t super_i = threadIdx.x + blockIdx.x * blockDim.x;
	if (super_i >= n_rays) return;

    uint32_t img = indice_image_for_tracking_pose;
    uint32_t half_kernel_window_size = kernel_window_size / 2;

	Eigen::Vector2i resolution = metadata[img].resolution;
	Eigen::Vector2i margins = Eigen::Vector2i(sample_away_from_border_margin_w, sample_away_from_border_margin_h);
	Eigen::Vector2i half_kernel_size = Eigen::Vector2i(half_kernel_window_size, half_kernel_window_size);

	rng.advance(super_i * N_MAX_RANDOM_SAMPLES_PER_RAY());
	Vector2f xy = nerf_random_image_pos_for_tracking(rng, resolution, snap_to_pixel_centers, margins, half_kernel_size);

    uint32_t base_i = super_i * ray_stride * 2;
    uint32_t cpt=0;
    Vector2f resolution_float = resolution.cast<float>();
    Vector2f half_kernel_size_float = half_kernel_size.cast<float>();
    // populate xy_image_pixel_indices with nearby indices  (within window)
	for (uint32_t u = 0; u < kernel_window_size; ++u) {
	    for (uint32_t v = 0; v < kernel_window_size; ++v) {

            Vector2i tmp_uv = Vector2i(v,u);
            Vector2f tmp_d = (tmp_uv.cast<float>() - half_kernel_size_float).cwiseQuotient(resolution_float);
            Vector2f tmp_xy = xy + tmp_d;
            xy_image_pixel_indices[base_i + cpt] = tmp_xy.x();
            xy_image_pixel_indices[base_i + cpt + 1] = tmp_xy.y();

            cpt+=2;
        }
    }
}

__global__ void generate_training_samples_for_tracking(
	const uint32_t n_rays,
	BoundingBox aabb,
	const uint32_t max_samples,
	default_rng_t rng,
	uint32_t* __restrict__ ray_counter,
	uint32_t* __restrict__ numsteps_counter,
	uint32_t* __restrict__ ray_indices_out,
	Ray* __restrict__ rays_out_unnormalized,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<NerfCoordinate> coords_out,
	const TrainingImageMetadata* __restrict__ metadata,
	const TrainingXForm* training_xforms,
	const uint8_t* __restrict__ density_grid,
	float cone_angle_constant,
	const float* __restrict__ distortion_data,
	const Vector2i distortion_resolution,
	const float* __restrict__ extra_dims_gpu,
	uint32_t n_extra_dims,
	const uint32_t indice_image_for_tracking_pose,
	int32_t* __restrict__ mapping_indices,
    const float* __restrict__ xy_image_pixel_indices
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_rays) return;

    uint32_t img = indice_image_for_tracking_pose;

	Eigen::Vector2i resolution = metadata[img].resolution;

	rng.advance(i * N_MAX_RANDOM_SAMPLES_PER_RAY());
    Vector2f xy = Vector2f(xy_image_pixel_indices[2*i], xy_image_pixel_indices[2*i+1]);

	mapping_indices[i] = -1; //  default to not existing

	// Negative values indicate masked-away regions
	size_t pix_idx = pixel_idx(xy, resolution, 0);
	if (read_rgba(xy, resolution, metadata[img].pixels, metadata[img].image_data_type).x() < 0.0f) {
		return;
	}

	float motionblur_time = 0.0;
	const Vector2f focal_length = metadata[img].focal_length;
	const Vector2f principal_point = metadata[img].principal_point;
	const float* extra_dims = extra_dims_gpu + img * n_extra_dims;
	const Lens lens = metadata[img].lens;

	const Matrix<float, 3, 4> xform = get_xform_given_rolling_shutter(training_xforms[img], metadata[img].rolling_shutter, xy, motionblur_time);

	Ray ray_unnormalized;
	const Ray* rays_in_unnormalized = metadata[img].rays;
	if (rays_in_unnormalized) {
		// Rays have been explicitly supplied. Read them.
		ray_unnormalized = rays_in_unnormalized[pix_idx];
	} else {
		// Rays need to be inferred from the camera matrix
		ray_unnormalized.o = xform.col(3);
		if (lens.mode == ELensMode::FTheta) {
			ray_unnormalized.d = f_theta_undistortion(xy - principal_point, lens.params, {0.f, 0.f, 1.f});
		} else if (lens.mode == ELensMode::LatLong) {
			ray_unnormalized.d = latlong_to_dir(xy);
		} else {
			ray_unnormalized.d = {
				(xy.x()-principal_point.x())*resolution.x() / focal_length.x(),
				(xy.y()-principal_point.y())*resolution.y() / focal_length.y(),
				1.0f,
			};

			if (lens.mode == ELensMode::OpenCV) {
				iterative_opencv_lens_undistortion(lens.params, &ray_unnormalized.d.x(), &ray_unnormalized.d.y());
			}
		}

		if (distortion_data) {
			ray_unnormalized.d.head<2>() += read_image<2>(distortion_data, distortion_resolution, xy);
		}

		ray_unnormalized.d = (xform.block<3, 3>(0, 0) * ray_unnormalized.d); // NOT normalized
	}

	Eigen::Vector3f ray_d_normalized = ray_unnormalized.d.normalized();

	Vector2f tminmax = aabb.ray_intersect(ray_unnormalized.o, ray_d_normalized);
	float cone_angle = calc_cone_angle(ray_d_normalized.dot(xform.col(2)), focal_length, cone_angle_constant);

	// The near distance prevents learning of camera-specific fudge right in front of the camera
	tminmax.x() = fmaxf(tminmax.x(), 0.0f);

	float startt = tminmax.x();
	startt += calc_dt(startt, cone_angle) * random_val(rng);
	Vector3f idir = ray_d_normalized.cwiseInverse();

	// first pass to compute an accurate number of steps
	uint32_t j = 0;
	float t=startt;
	Vector3f pos;

	while (aabb.contains(pos = ray_unnormalized.o + t * ray_d_normalized) && j < NERF_STEPS()) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray_d_normalized, idir, res);
		}
	}
	if (j == 0) {
		return;
	}
	uint32_t numsteps = j;
	uint32_t base = atomicAdd(numsteps_counter, numsteps);	 // first entry in the array is a counter
	if (base + numsteps > max_samples) {
		return;
	}

	coords_out += base;

	uint32_t ray_idx = atomicAdd(ray_counter, 1);

	ray_indices_out[ray_idx] = i;
	mapping_indices[i] = ray_idx;
	rays_out_unnormalized[ray_idx] = ray_unnormalized;
	numsteps_out[ray_idx*2+0] = numsteps;
	numsteps_out[ray_idx*2+1] = base;

	Vector3f warped_dir = warp_direction(ray_d_normalized);
	t=startt;
	j=0;
	while (aabb.contains(pos = ray_unnormalized.o + t * ray_d_normalized) && j < numsteps) {
		float dt = calc_dt(t, cone_angle);
		uint32_t mip = mip_from_dt(dt, pos);
		if (density_grid_occupied_at(pos, density_grid, mip)) {
			coords_out(j)->set_with_optional_extra_dims(warp_position(pos, aabb), warped_dir, warp_dt(dt), extra_dims, coords_out.stride_in_bytes);
			++j;
			t += dt;
		} else {
			uint32_t res = NERF_GRIDSIZE()>>mip;
			t = advance_to_next_voxel(t, cone_angle, pos, ray_d_normalized, idir, res);
		}
	}
}



__global__ void compute_GT_and_reconstructed_rgbd(
	const uint32_t n_rays,
	BoundingBox aabb,
	default_rng_t rng,
	const uint32_t target_batch_size,
	const uint32_t* __restrict__ rays_counter,
	int padded_output_width,
	const float* __restrict__ envmap_data,
	const Vector2i envmap_resolution,
	Array3f background_color,
	EColorSpace color_space,
	bool train_with_random_bg_color,
	bool train_in_linear_colors,
	const TrainingImageMetadata* __restrict__ metadata,
	const tcnn::network_precision_t* network_output,
	uint32_t* __restrict__ numsteps_counter,
	const uint32_t* __restrict__ ray_indices,
	const Ray* __restrict__ rays_in_unnormalized,
	uint32_t* __restrict__ numsteps_in,
	uint32_t* __restrict__ numsteps_out,
	PitchedPtr<const NerfCoordinate> coords_in,
	PitchedPtr<NerfCoordinate> coords_out,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	const Eigen::Array3f* __restrict__ exposure,
	float depth_supervision_lambda,
	const uint32_t indice_image_for_tracking_pose,
    const float* __restrict__ xy_image_pixel_indices,
	float* __restrict__ ground_truth_rgbd,
	float* __restrict__ reconstructed_rgbd
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_in[i*2+0];
	uint32_t base = numsteps_in[i*2+1];

	coords_in += base;
	network_output += base * padded_output_width;

	float T = 1.f;

	float EPSILON = 1e-4f;

	Array3f rgb_ray = Array3f::Zero();
	Vector3f hitpoint = Vector3f::Zero();
	float depth_ray = 0.f;

    Eigen::Vector3f ray_o = rays_in_unnormalized[i].o;

    uint32_t compacted_numsteps = 0;
	for (; compacted_numsteps < numsteps; ++compacted_numsteps) {
		if (T < EPSILON) {
			break;
		}

		const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		const Vector3f pos = unwarp_position(coords_in.ptr->pos.p, aabb);
		const float dt = unwarp_dt(coords_in.ptr->dt);
		float cur_depth = (pos - ray_o).norm();
		float density = network_to_density(float(local_network_output[3]), density_activation);


		const float alpha = 1.f - __expf(-density * dt);
		const float weight = alpha * T;
		rgb_ray += weight * rgb;
		hitpoint += weight * pos;
		depth_ray += weight * cur_depth;
		T *= (1.f - alpha);

		network_output += padded_output_width;
		coords_in += 1;
	}
	hitpoint /= (1.0f - T);

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices[i];
	rng.advance(ray_idx * N_MAX_RANDOM_SAMPLES_PER_RAY());

	uint32_t img = indice_image_for_tracking_pose;
	Eigen::Vector2i resolution = metadata[img].resolution;

    Vector2f xy = Vector2f(xy_image_pixel_indices[2*ray_idx], xy_image_pixel_indices[2*ray_idx+1]);

	if (train_with_random_bg_color) {
		background_color = random_val_3d(rng);
	}
	Array3f pre_envmap_background_color = background_color = srgb_to_linear(background_color);

	// Composit background behind envmap
	Array4f envmap_value;
	Vector3f dir;
	if (envmap_data) {
		dir = rays_in_unnormalized[i].d.normalized();
		envmap_value = read_envmap(envmap_data, envmap_resolution, dir);
		background_color = envmap_value.head<3>() + background_color * (1.0f - envmap_value.w());
	}

	Array3f exposure_scale = (0.6931471805599453f * exposure[img]).exp();

    Array4f texsamp = read_rgba(xy, resolution, metadata[img].pixels, metadata[img].image_data_type);

	Array3f rgbtarget;
	if (train_in_linear_colors || color_space == EColorSpace::Linear) {
		rgbtarget = exposure_scale * texsamp.head<3>() + (1.0f - texsamp.w()) * background_color;

		if (!train_in_linear_colors) {
			rgbtarget = linear_to_srgb(rgbtarget);
			background_color = linear_to_srgb(background_color);
		}
	} else if (color_space == EColorSpace::SRGB) {
		background_color = linear_to_srgb(background_color);
		if (texsamp.w() > 0) {
			rgbtarget = linear_to_srgb(exposure_scale * texsamp.head<3>() / texsamp.w()) * texsamp.w() + (1.0f - texsamp.w()) * background_color;
		} else {
			rgbtarget = background_color;
		}
	}

	if (compacted_numsteps == numsteps) {
		// support arbitrary background colors
		rgb_ray += T * background_color;
	}

    float target_depth = rays_in_unnormalized[i].d.norm() * ((depth_supervision_lambda > 0.0f && metadata[img].depth) ? read_depth(xy, resolution, metadata[img].depth) : -1.0f);

    ground_truth_rgbd[i*4+0] = rgbtarget.x();
    ground_truth_rgbd[i*4+1] = rgbtarget.y();
    ground_truth_rgbd[i*4+2] = rgbtarget.z();
    ground_truth_rgbd[i*4+3] = target_depth;

    reconstructed_rgbd[i*4+0] = rgb_ray.x();
    reconstructed_rgbd[i*4+1] = rgb_ray.y();
    reconstructed_rgbd[i*4+2] = rgb_ray.z();
    reconstructed_rgbd[i*4+3] = depth_ray;

	uint32_t compacted_base = atomicAdd(numsteps_counter, compacted_numsteps); // first entry in the array is a counter
	compacted_numsteps = min(target_batch_size - min(target_batch_size, compacted_base), compacted_numsteps);
	numsteps_out[i*2+0] = compacted_numsteps;
	numsteps_out[i*2+1] = compacted_base;

}


__global__ void compute_loss_and_gradient(
	const uint32_t n_rays,
	BoundingBox aabb,
	float loss_scale,
	int padded_output_width,
	const TrainingImageMetadata* __restrict__ metadata,
	const tcnn::network_precision_t* network_output,
	const uint32_t* __restrict__ ray_indices_in,
	const Ray* __restrict__ rays_in_unnormalized,
	uint32_t* __restrict__ numsteps_in,
	uint32_t* __restrict__ numsteps_compacted,
	PitchedPtr<const NerfCoordinate> coords_in,
	PitchedPtr<NerfCoordinate> coords_out,
	tcnn::network_precision_t* dloss_doutput,
	ELossType loss_type,
	ELossType depth_loss_type,
	float* __restrict__ loss_output,
	float* __restrict__ loss_depth_output,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	float* __restrict__ density_grid,
	const float* __restrict__ mean_density_ptr,
	float depth_supervision_lambda,
	float near_distance,
    const uint32_t ray_stride,
	const uint32_t kernel_window_size,
    float* __restrict__ xy_image_pixel_indices,
	const int32_t* __restrict__ mapping_indices,
	const float* __restrict__ ground_truth_rgbd,
	const float* __restrict__ reconstructed_rgbd,
	const uint32_t* __restrict__ ray_counter,
    const float* __restrict__ kernel
) {

	const uint32_t super_i = threadIdx.x + blockIdx.x * blockDim.x;
	if (super_i >= n_rays) { return; }


    // make gaussian kernel
    // MatrixXf kernel = make_gaussian_kernel(kernel_window_size, 11.f);

    // avg values within window
    float norm = 0.f;
    float norm_depth_target = 0.f;
	float avg_depth_ray = 0.f;
	Array3f avg_rgb_ray = Array3f::Zero();

    float avg_depth_ray_target = 0.f;
	Array3f avg_rgb_ray_target = Array3f::Zero();

    uint32_t i;
    uint32_t base_i = super_i*ray_stride;
    bool is_there_at_least_one_ray_in_super_ray=false;

    uint32_t cpt=0;
    for (uint32_t u = 0; u < kernel_window_size; ++u) {
	    for (uint32_t v = 0; v < kernel_window_size; ++v) {

            i = base_i + cpt;

            cpt++;

            int32_t ray_idx = mapping_indices[i];

            if (ray_idx < 0){
                //NOTE: if a ray is missing we can also discard the super ray.
                // ie break istead of continue
                continue;
            }

            is_there_at_least_one_ray_in_super_ray=true;

	        Array3f rgb_ray = {
                reconstructed_rgbd[4*ray_idx],
                reconstructed_rgbd[4*ray_idx+1],
                reconstructed_rgbd[4*ray_idx+2],
            };

            float depth_ray = reconstructed_rgbd[4*ray_idx+3];

            avg_rgb_ray += kernel[cpt] * rgb_ray;
            avg_depth_ray += kernel[cpt] * depth_ray;

	        Array3f rgb_ray_target = {
                ground_truth_rgbd[4*ray_idx],
                ground_truth_rgbd[4*ray_idx+1],
                ground_truth_rgbd[4*ray_idx+2],
            };

            float depth_ray_target = ground_truth_rgbd[4*ray_idx+3];

            avg_rgb_ray_target += kernel[cpt] * rgb_ray_target;

            // handle cases where depth is 0.0 or -1.0
            if (depth_ray_target > 0.0) {
                avg_depth_ray_target += kernel[cpt] * depth_ray_target;
                norm_depth_target += kernel[cpt];
            }

            norm += kernel[cpt];
        }
    }

    //If all rays in super ray have 0 numsteps
    if (!is_there_at_least_one_ray_in_super_ray){
        return;
    }

    avg_rgb_ray /= norm;
    avg_depth_ray /= norm;

    avg_rgb_ray_target /= norm;
    avg_depth_ray_target /= norm_depth_target;

	// Step again, this time computing loss
    LossAndGradient lg = loss_and_gradient(avg_rgb_ray_target, avg_rgb_ray, loss_type);
	LossAndGradient lg_depth = loss_and_gradient(Array3f::Constant(avg_depth_ray_target), Array3f::Constant(avg_depth_ray), depth_loss_type);

    float depth_loss_gradient = avg_depth_ray_target > 0.0f ? depth_supervision_lambda * lg_depth.gradient.x() : 0;

    uint32_t total_n_rays = *ray_counter;

    float mean_loss = lg.loss.mean();
	if (loss_output) {
        cpt=0;
	    for (uint32_t u = 0; u < kernel_window_size; ++u) {
	        for (uint32_t v = 0; v < kernel_window_size; ++v) {
                i = base_i + cpt;
                cpt++;
                int32_t ray_idx = mapping_indices[i];
                if (ray_idx < 0){
		            loss_output[i] = 0.f;
                } else {
		            loss_output[i] = mean_loss / (float)total_n_rays;
                }
            }
        }
	}
	if (loss_depth_output) {
        cpt=0;
	    for (uint32_t u = 0; u < kernel_window_size; ++u) {
	        for (uint32_t v = 0; v < kernel_window_size; ++v) {
                i = base_i + cpt;
                cpt++;
                int32_t ray_idx = mapping_indices[i];
                if (ray_idx < 0){
		            loss_depth_output[i] = 0.f;
                } else {
                    float depth_ray_target = ground_truth_rgbd[4*ray_idx+3];
                    if (depth_ray_target>0.0) {
		                loss_depth_output[i] = lg_depth.loss.x() / (float)total_n_rays;
                    } else {
		                loss_depth_output[i] = 0.f;
                    }
                }
            }
        }
	}


    loss_scale /= total_n_rays;

    // No regularization for pose optimization
	const float output_l2_reg = 0.0f;
	const float output_l1_reg_density = 0.0f;

	// now do it again computing gradients
    cpt=0;
	for (uint32_t u = 0; u < kernel_window_size; ++u) {
	    for (uint32_t v = 0; v < kernel_window_size; ++v) {
            i = base_i + cpt;
            cpt++;

            int32_t ray_idx = mapping_indices[i];

            if (ray_idx < 0){
                continue;
            }

            Array3f rgb_ray = {
                reconstructed_rgbd[4*ray_idx],
                reconstructed_rgbd[4*ray_idx+1],
                reconstructed_rgbd[4*ray_idx+2],
            };
            float depth_ray = reconstructed_rgbd[4*ray_idx+3];
            float depth_ray_target = ground_truth_rgbd[4*ray_idx+3];

            Array3f rgb_ray2 = { 0.f,0.f,0.f };
	        float depth_ray2 = 0.f;
	        float T = 1.f;

            uint32_t base = numsteps_in[ray_idx*2+1];
	        uint32_t base_compact = numsteps_compacted[ray_idx*2+1];
	        uint32_t numsteps_compact = numsteps_compacted[ray_idx*2];

            coords_out += base_compact;
	        dloss_doutput += base_compact * padded_output_width;

	        coords_in += base;
	        network_output += base * padded_output_width;

	        Eigen::Vector3f ray_o = rays_in_unnormalized[ray_idx].o;

            for (uint32_t j=0; j < numsteps_compact; ++j) {

                // Compact network inputs
		        NerfCoordinate* coord_out = coords_out(j);
		        const NerfCoordinate* coord_in = coords_in(j);
		        coord_out->copy(*coord_in, coords_out.stride_in_bytes);

		        const Vector3f pos = unwarp_position(coord_in->pos.p, aabb);
		        float depth = (pos - ray_o).norm();

		        float dt = unwarp_dt(coord_in->dt);
		        const tcnn::vector_t<tcnn::network_precision_t, 4> local_network_output = *(tcnn::vector_t<tcnn::network_precision_t, 4>*)network_output;
		        const Array3f rgb = network_to_rgb(local_network_output, rgb_activation);
		        const float density = network_to_density(float(local_network_output[3]), density_activation);
		        const float alpha = 1.f - __expf(-density * dt);
		        const float weight = alpha * T;
		        rgb_ray2 += weight * rgb;
		        depth_ray2 += weight * depth;
		        T *= (1.f - alpha);


		        // we know the suffix of this ray compared to where we are up to. note the suffix depends on this step's alpha as suffix = (1-alpha)*(somecolor), so dsuffix/dalpha = -somecolor = -suffix/(1-alpha)
		        const Array3f suffix = rgb_ray - rgb_ray2;
		        const Array3f dloss_by_drgb = weight * lg.gradient * kernel[cpt] / norm;

		        tcnn::vector_t<tcnn::network_precision_t, 4> local_dL_doutput;

		        // chain rule to go from dloss/drgb to dloss/dmlp_output
                local_dL_doutput[0] = loss_scale * (dloss_by_drgb.x() * network_to_rgb_derivative(local_network_output[0], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[0])); // Penalize way too large color values
		        local_dL_doutput[1] = loss_scale * (dloss_by_drgb.y() * network_to_rgb_derivative(local_network_output[1], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[1]));
		        local_dL_doutput[2] = loss_scale * (dloss_by_drgb.z() * network_to_rgb_derivative(local_network_output[2], rgb_activation) + fmaxf(0.0f, output_l2_reg * (float)local_network_output[2]));

		        float density_derivative = network_to_density_derivative(float(local_network_output[3]), density_activation);
		        const float depth_suffix = depth_ray - depth_ray2;

                // if no target depth for that ray then no depth supervision
                float depth_supervision = 0.0f;
                if (depth_ray_target > 0.0f) {
		            depth_supervision = depth_loss_gradient * (kernel[cpt] / norm) * (T * depth - depth_suffix);
                }

		        float dloss_by_dmlp = density_derivative * (
		        	dt * (lg.gradient.matrix().dot((T * rgb - suffix).matrix()) + depth_supervision)
		        );

		        local_dL_doutput[3] =
		        	loss_scale * dloss_by_dmlp +
		        	(float(local_network_output[3]) < 0.0f ? -output_l1_reg_density : 0.0f) +
		        	(float(local_network_output[3]) > -10.0f && depth < near_distance ? 1e-4f : 0.0f);
		        	;


		        *(tcnn::vector_t<tcnn::network_precision_t, 4>*)dloss_doutput = local_dL_doutput;

		        dloss_doutput += padded_output_width;
		        network_output += padded_output_width;
            }

		    dloss_doutput -= numsteps_compact*padded_output_width;
		    network_output -= numsteps_compact*padded_output_width;

            coords_in -= base;
	        network_output -= base * padded_output_width;

            coords_out -= base_compact;
	        dloss_doutput -= base_compact * padded_output_width;

        }
	}
}



__global__ void compute_camera_gradient(
	const uint32_t n_rays,
	const BoundingBox aabb,
	const uint32_t* __restrict__ rays_counter,
	const TrainingXForm* training_xforms,
	Vector3f* cam_pos_gradient,
	Vector3f* cam_rot_gradient,
	const TrainingImageMetadata* __restrict__ metadata,
	const uint32_t* __restrict__ ray_indices,
	const Ray* __restrict__ rays_in_unnormalized,
	uint32_t* __restrict__ numsteps_compacted,
	PitchedPtr<NerfCoordinate> coords,
	PitchedPtr<NerfCoordinate> coords_gradient,
	float* __restrict__ distortion_gradient,
	float* __restrict__ distortion_gradient_weight,
	const Vector2i distortion_resolution,
	Vector2f* cam_focal_length_gradient,
	const uint32_t indice_image_for_tracking_pose,
    float* __restrict__ xy_image_pixel_indices
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= *rays_counter) { return; }

	// grab the number of samples for this ray, and the first sample
	uint32_t numsteps = numsteps_compacted[i*2];
	if (numsteps == 0) {
		// The ray doesn't matter. So no gradient onto the camera
		return;
	}

	uint32_t base = numsteps_compacted[i*2+1];
	coords += base;
	coords_gradient += base;

	// Must be same seed as above to obtain the same
	// background color.
	uint32_t ray_idx = ray_indices[i];
    uint32_t img = indice_image_for_tracking_pose;

	Eigen::Vector2i resolution = metadata[img].resolution;

	const Matrix<float, 3, 4>& xform = training_xforms[img].start;

	Ray ray = rays_in_unnormalized[i];
	ray.d = ray.d.normalized();
	Ray ray_gradient = { Vector3f::Zero(), Vector3f::Zero() };

	// Compute ray gradient
	for (uint32_t j = 0; j < numsteps; ++j) {
		// pos = ray.o + t * ray.d;

		const Vector3f warped_pos = coords(j)->pos.p;
		const Vector3f pos_gradient = coords_gradient(j)->pos.p.cwiseProduct(warp_position_derivative(warped_pos, aabb));
		ray_gradient.o += pos_gradient;
		const Vector3f pos = unwarp_position(warped_pos, aabb);

		// Scaled by t to account for the fact that further-away objects' position
		// changes more rapidly as the direction changes.
		float t = (pos - ray.o).norm();
		const Vector3f dir_gradient = coords_gradient(j)->dir.d.cwiseProduct(warp_direction_derivative(coords(j)->dir.d));
		ray_gradient.d += pos_gradient * t + dir_gradient;
	}

	float xy_pdf = 1.0f;
    Vector2f xy = Vector2f(xy_image_pixel_indices[2*ray_idx], xy_image_pixel_indices[2*ray_idx+1]);

	if (distortion_gradient) {
		// Projection of the raydir gradient onto the plane normal to raydir,
		// because that's the only degree of motion that the raydir has.
		Vector3f orthogonal_ray_gradient = ray_gradient.d - ray.d * ray_gradient.d.dot(ray.d);

		// Rotate ray gradient to obtain image plane gradient.
		// This has the effect of projecting the (already projected) ray gradient from the
		// tangent plane of the sphere onto the image plane (which is correct!).
		Vector3f image_plane_gradient = xform.block<3,3>(0,0).inverse() * orthogonal_ray_gradient;

		// Splat the resulting 2D image plane gradient into the distortion params
		deposit_image_gradient<2>(image_plane_gradient.head<2>() / xy_pdf, distortion_gradient, distortion_gradient_weight, distortion_resolution, xy);
	}

	if (cam_pos_gradient) {
		// Atomically reduce the ray gradient into the xform gradient
		NGP_PRAGMA_UNROLL
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_pos_gradient[img][j], ray_gradient.o[j] / xy_pdf);
		}
	}

	if (cam_rot_gradient) {
		// Rotation is averaged in log-space (i.e. by averaging angle-axes).
		// Due to our construction of ray_gradient.d, ray_gradient.d and ray.d are
		// orthogonal, leading to the angle_axis magnitude to equal the magnitude
		// of ray_gradient.d.
		Vector3f angle_axis = ray.d.cross(ray_gradient.d);

		// Atomically reduce the ray gradient into the xform gradient
		NGP_PRAGMA_UNROLL
		for (uint32_t j = 0; j < 3; ++j) {
			atomicAdd(&cam_rot_gradient[img][j], angle_axis[j] / xy_pdf);
		}
	}
}




void Testbed::track_pose_nerf_slam_opti(uint32_t target_batch_size, bool get_loss_scalar, hipStream_t stream) {

	if (m_nerf.training.indice_image_for_tracking_pose == 0) {
        // no tracking for first frame.
		return;
	}

	m_nerf.training.counters_rgb_track.prepare_for_training_steps(stream);

	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_pos_gradient_gpu.data(), 0, m_nerf.training.cam_pos_gradient_gpu.get_bytes(), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_nerf.training.cam_rot_gradient_gpu.data(), 0, m_nerf.training.cam_rot_gradient_gpu.get_bytes(), stream));

	track_pose_nerf_slam_step_opti(target_batch_size, m_nerf.training.counters_rgb_track, stream);

    ++m_training_step_track;

    std::vector<float> losses_scalar = m_nerf.training.counters_rgb_track.update_after_training(target_batch_size, get_loss_scalar, stream, true);
    float loss_scalar = losses_scalar[0];
    float loss_depth_scalar = losses_scalar[1];
	bool zero_records = m_nerf.training.counters_rgb_track.measured_batch_size == 0;
	if (get_loss_scalar) {
        m_tracking_loss = loss_scalar;
        m_tracking_loss_depth = loss_depth_scalar;
		m_loss_scalar_track.update(loss_scalar);
	}

	if (zero_records) {
		m_loss_scalar_track.set(0.f);
		tlog::warning() << "Nerf training generated 0 samples. Aborting training.";
		m_train = false;
	}

	// Get extrinsics gradients
    {
		// float per_camera_loss_scale = (float)m_nerf.training.n_images_for_training / LOSS_SCALE / (float)m_nerf.training.n_steps_between_cam_updates;
		float per_camera_loss_scale = 1.0 / LOSS_SCALE;

		{
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_pos_gradient.data(), m_nerf.training.cam_pos_gradient_gpu.data(), m_nerf.training.cam_pos_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipMemcpyAsync(m_nerf.training.cam_rot_gradient.data(), m_nerf.training.cam_rot_gradient_gpu.data(), m_nerf.training.cam_rot_gradient_gpu.get_bytes(), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));

			// Optimization step
            uint32_t i = m_nerf.training.indice_image_for_tracking_pose;
			Vector3f pos_gradient = m_nerf.training.cam_pos_gradient[i] * per_camera_loss_scale;
			Vector3f rot_gradient = m_nerf.training.cam_rot_gradient[i] * per_camera_loss_scale;

			float l2_reg = m_nerf.training.extrinsic_l2_reg;
			pos_gradient += m_nerf.training.cam_pos_offset[i].variable() * l2_reg;
			rot_gradient += m_nerf.training.cam_rot_offset[i].variable() * l2_reg;

            if (m_nerf.training.separate_pos_and_rot_lr) {
			    m_nerf.training.cam_pos_offset[i].set_learning_rate(m_nerf.training.extrinsic_learning_rate_pos);
			    m_nerf.training.cam_rot_offset[i].set_learning_rate(m_nerf.training.extrinsic_learning_rate_rot);
            } else {
			    m_nerf.training.cam_pos_offset[i].set_learning_rate(m_nerf.training.extrinsic_learning_rate);
			    m_nerf.training.cam_rot_offset[i].set_learning_rate(m_nerf.training.extrinsic_learning_rate);
            }

            tlog::info()<<" pos gradient = "<< pos_gradient[0]<<", "<< pos_gradient[1]<<", "<<pos_gradient[2];

			m_nerf.training.cam_pos_offset[i].step(pos_gradient);
			m_nerf.training.cam_rot_offset[i].step(rot_gradient);

			m_nerf.training.update_transforms(i, i+1);
		}
	}
}


void Testbed::track_pose_nerf_slam_step_opti(uint32_t target_batch_size, Testbed::NerfCounters& counters, hipStream_t stream) {
	const uint32_t padded_output_width = m_network->padded_output_width();
	const uint32_t max_samples = target_batch_size * 16; // Somewhat of a worst case
	const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + m_nerf_network->n_extra_dims();
	const uint32_t extra_stride = m_nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

	GPUMemoryArena::Allocation alloc;
	auto scratch = allocate_workspace_and_distribute<
		uint32_t, // ray_indices
		Ray, // rays
		uint32_t, // numsteps
		float, // coords
		float, // max_level
		network_precision_t, // mlp_out
		network_precision_t, // dloss_dmlp_out
		float, // coords_compacted
		float, // coords_gradient
		float, // max_level_compacted
		uint32_t, // ray_counter
		float, // xy_pixel_indices
		int32_t, // mapping_indices
		float, // ground_truth_rgbd
		float,  // reconstructed_rgbd
		uint32_t // numsteps_compacted
	>(
		stream, &alloc,
		counters.rays_per_batch,
		counters.rays_per_batch,
		counters.rays_per_batch * 2,
		max_samples * floats_per_coord,
		max_samples,
		std::max(target_batch_size, max_samples) * padded_output_width,
		target_batch_size * padded_output_width,
		target_batch_size * floats_per_coord,
		target_batch_size * floats_per_coord,
		target_batch_size,
		1,
		counters.rays_per_batch * 2,
		counters.rays_per_batch,
		counters.rays_per_batch * 4,
		counters.rays_per_batch * 4,
		counters.rays_per_batch * 2
	);

	// TODO: C++17 structured binding
	uint32_t* ray_indices = std::get<0>(scratch);
	Ray* rays_unnormalized = std::get<1>(scratch);
	uint32_t* numsteps = std::get<2>(scratch);
	float* coords = std::get<3>(scratch);
	float* max_level = std::get<4>(scratch);
	network_precision_t* mlp_out = std::get<5>(scratch);
	network_precision_t* dloss_dmlp_out = std::get<6>(scratch);
	float* coords_compacted = std::get<7>(scratch);
	float* coords_gradient = std::get<8>(scratch);
	float* max_level_compacted = std::get<9>(scratch);
	uint32_t* ray_counter = std::get<10>(scratch);
	float* xy_image_pixel_indices = std::get<11>(scratch);
	int32_t* mapping_indices = std::get<12>(scratch);
	float* ground_truth_rgbd = std::get<13>(scratch);
	float* reconstructed_rgbd = std::get<14>(scratch);
	uint32_t* numsteps_compacted = std::get<15>(scratch);

	uint32_t max_inference;
	if (counters.measured_batch_size_before_compaction == 0) {
		counters.measured_batch_size_before_compaction = max_inference = max_samples;
	} else {
		max_inference = next_multiple(std::min(counters.measured_batch_size_before_compaction, max_samples), tcnn::batch_size_granularity);
	}

	GPUMatrix<float> coords_matrix((float*)coords, floats_per_coord, max_inference);
	GPUMatrix<network_precision_t> rgbsigma_matrix(mlp_out, padded_output_width, max_inference);

	GPUMatrix<float> compacted_coords_matrix((float*)coords_compacted, floats_per_coord, target_batch_size);
	GPUMatrix<network_precision_t> compacted_rgbsigma_matrix(mlp_out, padded_output_width, target_batch_size);

	GPUMatrix<network_precision_t> gradient_matrix(dloss_dmlp_out, padded_output_width, target_batch_size);

	if (m_training_step_track == 0) {
		counters.n_rays_total = 0;
	}

	counters.n_rays_total += counters.rays_per_batch;
	m_nerf.training.n_rays_since_error_map_update += counters.rays_per_batch;

    //NOTE: get settings/hyperparams for tracking
    const float sigma = m_tracking_sigma_gaussian_kernel;
    const uint32_t kernel_window_size = m_tracking_kernel_window_size;
    uint32_t ray_stride = kernel_window_size*kernel_window_size;
    uint32_t sample_away_from_border_margin_h = m_sample_away_from_border_margin_h;
    uint32_t sample_away_from_border_margin_w = m_sample_away_from_border_margin_w;
    uint32_t n_super_rays = counters.rays_per_batch / ray_stride; // get the number of rays for which we have enough room to get the corresponding nearby rays (within window)

    m_track_pose_nerf_num_super_rays_targeted_in_tracking_step=n_super_rays;

	CUDA_CHECK_THROW(hipMemsetAsync(ray_counter, 0, sizeof(uint32_t), stream));

    // create gaussian kernel
    std::vector<float> kernel = make_gaussian_kernel_debug(kernel_window_size, sigma);

    tcnn::GPUMemory<float> kernel_gpu;
    kernel_gpu.enlarge(kernel_window_size * kernel_window_size);

    CUDA_CHECK_THROW(
       hipMemcpy(
          kernel_gpu.data(),
          kernel.data(),
          kernel_window_size * kernel_window_size * sizeof(float),
          hipMemcpyHostToDevice
       )
    );


    //NOTE: get sample xy pixel locations
    linear_kernel(sample_training_pixels_for_tracking, 0, stream,
		n_super_rays,
        ray_stride,
		m_rng,
        kernel_window_size,
        sample_away_from_border_margin_h,
        sample_away_from_border_margin_w,
		m_nerf.training.dataset.metadata_gpu.data(),
        m_nerf.training.indice_image_for_tracking_pose,
		m_nerf.training.snap_to_pixel_centers,
        xy_image_pixel_indices
	);

    //NOTE: get sample along each rays
	linear_kernel(generate_training_samples_for_tracking, 0, stream,
		n_super_rays*ray_stride,
		m_aabb,
		max_inference,
		m_rng,
		ray_counter,
		counters.numsteps_counter.data(),
		ray_indices,
		rays_unnormalized,
		numsteps,
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
        m_nerf.training.dataset.metadata_gpu.data(),
		m_nerf.training.transforms_gpu.data(),
		m_nerf.density_grid_bitfield.data(),
		m_nerf.cone_angle_constant,
		m_distortion.map->params(),
		m_distortion.resolution,
		m_nerf.training.extra_dims_gpu.data(),
		m_nerf_network->n_extra_dims(),
        m_nerf.training.indice_image_for_tracking_pose,
		mapping_indices,
        xy_image_pixel_indices
	);


	CUDA_CHECK_THROW(
       hipMemcpyAsync(
          &m_track_pose_nerf_num_rays_in_tracking_step,
          std::get<10>(scratch),
          sizeof(uint32_t),
          hipMemcpyDeviceToHost,
          stream
       )
    );


    //NOTE: get network values for each points
	m_network->inference_mixed_precision(stream, coords_matrix, rgbsigma_matrix, false);


    //NOTE: get RGBD values prediciton + GT.
	linear_kernel(compute_GT_and_reconstructed_rgbd, 0, stream,
		counters.rays_per_batch,
		m_aabb,
		m_rng,
		target_batch_size,
		ray_counter,
		padded_output_width,
		m_envmap.envmap->params(),
		m_envmap.resolution,
		m_background_color.head<3>(),
		m_color_space,
		m_nerf.training.random_bg_color,
		m_nerf.training.linear_colors,
		m_nerf.training.dataset.metadata_gpu.data(),
		mlp_out,
		counters.numsteps_counter_compacted.data(),
		ray_indices,
		rays_unnormalized,
		numsteps,
		numsteps_compacted,
		PitchedPtr<const NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1 ,0, extra_stride),
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.cam_exposure_gpu.data(),
		m_nerf.training.depth_supervision_lambda,
        m_nerf.training.indice_image_for_tracking_pose,
        xy_image_pixel_indices,
        ground_truth_rgbd,
        reconstructed_rgbd
	);


    //NOTE: compute loss and gradients.
	linear_kernel(compute_loss_and_gradient, 0, stream,
		n_super_rays,
		m_aabb,
		LOSS_SCALE,
		padded_output_width,
		m_nerf.training.dataset.metadata_gpu.data(),
		mlp_out,
		ray_indices,
		rays_unnormalized,
		numsteps,
        numsteps_compacted,
		PitchedPtr<const NerfCoordinate>((NerfCoordinate*)coords, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1 ,0, extra_stride),
		dloss_dmlp_out,
		m_nerf.training.track_loss_type,
		m_nerf.training.track_depth_loss_type,
		counters.loss.data(),
		counters.loss_depth.data(),
		m_nerf.rgb_activation,
		m_nerf.density_activation,
		m_nerf.density_grid.data(),
		m_nerf.density_grid_mean.data(),
		m_nerf.training.depth_supervision_lambda,
		m_nerf.training.near_distance,
        ray_stride,
        kernel_window_size,
        xy_image_pixel_indices,
		mapping_indices,
        ground_truth_rgbd,
        reconstructed_rgbd,
		ray_counter,
        kernel_gpu.data()
	);

	fill_rollover_and_rescale<network_precision_t><<<n_blocks_linear(target_batch_size*padded_output_width), n_threads_linear, 0, stream>>>(
		target_batch_size, padded_output_width, counters.numsteps_counter_compacted.data(), dloss_dmlp_out
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size * floats_per_coord), n_threads_linear, 0, stream>>>(
		target_batch_size, floats_per_coord, counters.numsteps_counter_compacted.data(), (float*)coords_compacted
	);
	fill_rollover<float><<<n_blocks_linear(target_batch_size), n_threads_linear, 0, stream>>>(
		target_batch_size, 1, counters.numsteps_counter_compacted.data(), max_level_compacted
	);

	bool train_camera = true;
	bool prepare_input_gradients = train_camera;
	GPUMatrix<float> coords_gradient_matrix((float*)coords_gradient, floats_per_coord, target_batch_size);

	{
		auto ctx = m_network->forward(stream, compacted_coords_matrix, &compacted_rgbsigma_matrix, false, prepare_input_gradients);
		m_network->backward(stream, *ctx, compacted_coords_matrix, compacted_rgbsigma_matrix, gradient_matrix, prepare_input_gradients ? &coords_gradient_matrix : nullptr, false, EGradientMode::Overwrite);
	}

	// Compute camera gradients
	linear_kernel(compute_camera_gradient, 0, stream,
		counters.rays_per_batch,
		m_aabb,
		ray_counter,
		m_nerf.training.transforms_gpu.data(),
		m_nerf.training.cam_pos_gradient_gpu.data(),
		m_nerf.training.cam_rot_gradient_gpu.data(),
		m_nerf.training.dataset.metadata_gpu.data(),
		ray_indices,
		rays_unnormalized,
		numsteps_compacted,
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_compacted, 1, 0, extra_stride),
		PitchedPtr<NerfCoordinate>((NerfCoordinate*)coords_gradient, 1, 0, extra_stride),
		m_nerf.training.optimize_distortion ? m_distortion.map->gradients() : nullptr,
		m_nerf.training.optimize_distortion ? m_distortion.map->gradient_weights() : nullptr,
		m_distortion.resolution,
		m_nerf.training.optimize_focal_length ? m_nerf.training.cam_focal_length_gradient_gpu.data() : nullptr,
        m_nerf.training.indice_image_for_tracking_pose,
        xy_image_pixel_indices
	);

	m_rng.advance();

}


NGP_NAMESPACE_END
