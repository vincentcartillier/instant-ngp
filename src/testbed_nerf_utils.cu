#include "hip/hip_runtime.h"

#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>

#ifdef copysign
#undef copysign
#endif

using namespace Eigen;
using namespace tcnn;
namespace fs = filesystem;

NGP_NAMESPACE_BEGIN


inline constexpr __device__ float NERF_RENDERING_NEAR_DISTANCE() { return 0.05f; }
inline constexpr __device__ uint32_t NERF_STEPS() { return 1024; } // finest number of steps per unit length
inline constexpr __device__ uint32_t NERF_CASCADES() { return 8; }

inline constexpr __device__ float SQRT3() { return 1.73205080757f; }
inline constexpr __device__ float STEPSIZE() { return (SQRT3() / NERF_STEPS()); } // for nerf raymarch
inline constexpr __device__ float MIN_CONE_STEPSIZE() { return STEPSIZE(); }
// Maximum step size is the width of the coarsest gridsize cell.
inline constexpr __device__ float MAX_CONE_STEPSIZE() { return STEPSIZE() * (1<<(NERF_CASCADES()-1)) * NERF_STEPS() / NERF_GRIDSIZE(); }

// Used to index into the PRNG stream. Must be larger than the number of
// samples consumed by any given training ray.
inline constexpr __device__ uint32_t N_MAX_RANDOM_SAMPLES_PER_RAY() { return 8; }

// Any alpha below this is considered "invisible" and is thus culled away.
inline constexpr __device__ float NERF_MIN_OPTICAL_THICKNESS() { return 0.01f; }

static constexpr uint32_t MARCH_ITER = 10000;

static constexpr uint32_t MIN_STEPS_INBETWEEN_COMPACTION = 1;
static constexpr uint32_t MAX_STEPS_INBETWEEN_COMPACTION = 8;

static constexpr float UNIFORM_SAMPLING_FRACTION = 0.5f;


struct LossAndGradient {
	vec3 loss;
	vec3 gradient;

	__host__ __device__ LossAndGradient operator*(float scalar) {
		return {loss * scalar, gradient * scalar};
	}

	__host__ __device__ LossAndGradient operator/(float scalar) {
		return {loss / scalar, gradient / scalar};
	}
};

inline __device__ Vector2f sample_cdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, float* __restrict__ pdf) {
	if (sample.x() < UNIFORM_SAMPLING_FRACTION) {
		sample.x() /= UNIFORM_SAMPLING_FRACTION;
		return sample;
	}

	sample.x() = (sample.x() - UNIFORM_SAMPLING_FRACTION) / (1.0f - UNIFORM_SAMPLING_FRACTION);

	cdf_y += img * res.y();

	// First select row according to cdf_y
	uint32_t y = binary_search(sample.y(), cdf_y, res.y());
	float prev = y > 0 ? cdf_y[y-1] : 0.0f;
	float pmf_y = cdf_y[y] - prev;
	sample.y() = (sample.y() - prev) / pmf_y;

	cdf_x_cond_y += img * res.y() * res.x() + y * res.x();

	// Then, select col according to x
	uint32_t x = binary_search(sample.x(), cdf_x_cond_y, res.x());
	prev = x > 0 ? cdf_x_cond_y[x-1] : 0.0f;
	float pmf_x = cdf_x_cond_y[x] - prev;
	sample.x() = (sample.x() - prev) / pmf_x;

	if (pdf) {
		*pdf = pmf_x * pmf_y * res.prod();
	}

	return {((float)x + sample.x()) / (float)res.x(), ((float)y + sample.y()) / (float)res.y()};
}



inline __device__ Vector2f nerf_random_image_pos_training(default_rng_t& rng, const Vector2i& resolution, bool snap_to_pixel_centers, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y, const Vector2i& cdf_res, uint32_t img, float* __restrict__ pdf = nullptr) {
	Vector2f xy = random_val_2d(rng);

	if (cdf_x_cond_y) {
		xy = sample_cdf_2d(xy, img, cdf_res, cdf_x_cond_y, cdf_y, pdf);
	} else if (pdf) {
		*pdf = 1.0f;
	}

	if (snap_to_pixel_centers) {
		xy = (xy.cwiseProduct(resolution.cast<float>()).cast<int>().cwiseMax(0).cwiseMin(resolution - Vector2i::Ones()).cast<float>() + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	}
	return xy;
}


inline __host__ __device__ uint32_t grid_mip_offset(uint32_t mip) {
	return NERF_GRID_N_CELLS() * mip;
}

inline __host__ __device__ float calc_cone_angle(float cosine, const Eigen::Vector2f& focal_length, float cone_angle_constant) {
	// Pixel size. Doesn't always yield a good performance vs. quality
	// trade off. Especially if training pixels have a much different
	// size than rendering pixels.
	// return cosine*cosine / focal_length.mean();

	return cone_angle_constant;
}

inline __host__ __device__ float to_stepping_space(float t, float cone_angle) {
	if (cone_angle <= 1e-5f) {
		return t / MIN_CONE_STEPSIZE();
	}

	float log1p_c = logf(1.0f + cone_angle);

	float a = (logf(MIN_CONE_STEPSIZE()) - logf(log1p_c)) / log1p_c;
	float b = (logf(MAX_CONE_STEPSIZE()) - logf(log1p_c)) / log1p_c;

	float at = expf(a * log1p_c);
	float bt = expf(b * log1p_c);

	if (t <= at) {
		return (t - at) / MIN_CONE_STEPSIZE() + a;
	} else if (t <= bt) {
		return logf(t) / log1p_c;
	} else {
		return (t - bt) / MAX_CONE_STEPSIZE() + b;
	}
}

inline __host__ __device__ float from_stepping_space(float n, float cone_angle) {
	if (cone_angle <= 1e-5f) {
		return n * MIN_CONE_STEPSIZE();
	}

	float log1p_c = logf(1.0f + cone_angle);

	float a = (logf(MIN_CONE_STEPSIZE()) - logf(log1p_c)) / log1p_c;
	float b = (logf(MAX_CONE_STEPSIZE()) - logf(log1p_c)) / log1p_c;

	float at = expf(a * log1p_c);
	float bt = expf(b * log1p_c);

	if (n <= a) {
		return (n - a) * MIN_CONE_STEPSIZE() + at;
	} else if (n <= b) {
		return expf(n * log1p_c);
	} else {
		return (n - b) * MAX_CONE_STEPSIZE() + bt;
	}
}



inline __host__ __device__ float advance_n_steps(float t, float cone_angle, float n) {
	return from_stepping_space(to_stepping_space(t, cone_angle) + n, cone_angle);
}


inline __host__ __device__ float calc_dt(float t, float cone_angle) {
	return advance_n_steps(t, cone_angle, 1.0f) - t;
}

inline __device__ vec3 copysign(const vec3& a, const vec3& b) {
	return {
		copysignf(a.x, b.x),
		copysignf(a.y, b.y),
		copysignf(a.z, b.z),
	};
}

inline __device__ LossAndGradient l2_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	return {
		difference * difference,
		2.0f * difference
	};
}



inline __device__ LossAndGradient relative_l2_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	vec3 denom = prediction * prediction + vec3(1e-2f);
	return {
		difference * difference / denom,
		2.0f * difference / denom
	};
}


inline __device__ LossAndGradient l1_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	return {
		abs(difference),
		copysign(vec3(1.0f), difference),
	};
}


inline __device__ LossAndGradient huber_loss(const vec3& target, const vec3& prediction, float alpha = 1) {
	vec3 difference = prediction - target;
	vec3 abs_diff = abs(difference);
	vec3 square = 0.5f/alpha * difference * difference;
	return {
		{
			abs_diff.x > alpha ? (abs_diff.x - 0.5f * alpha) : square.x,
			abs_diff.y > alpha ? (abs_diff.y - 0.5f * alpha) : square.y,
			abs_diff.z > alpha ? (abs_diff.z - 0.5f * alpha) : square.z,
		},
		{
			abs_diff.x > alpha ? (difference.x > 0 ? 1.0f : -1.0f) : (difference.x / alpha),
			abs_diff.y > alpha ? (difference.y > 0 ? 1.0f : -1.0f) : (difference.y / alpha),
			abs_diff.z > alpha ? (difference.z > 0 ? 1.0f : -1.0f) : (difference.z / alpha),
		},
	};
}








inline __device__ LossAndGradient log_l1_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	vec3 divisor = abs(difference) + vec3(1.0f);
	return {
		log(divisor),
		copysign(vec3(1.0f) / divisor, difference),
	};
}

inline __device__ LossAndGradient smape_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	vec3 denom = 0.5f * (abs(prediction) + abs(target)) + vec3(1e-2f);
	return {
		abs(difference) / denom,
		copysign(vec3(1.0f) / denom, difference),
	};
}

inline __device__ LossAndGradient mape_loss(const vec3& target, const vec3& prediction) {
	vec3 difference = prediction - target;
	vec3 denom = abs(prediction) + vec3(1e-2f);
	return {
		abs(difference) / denom,
		copysign(vec3(1.0f) / denom, difference),
	};
}



inline __device__ float distance_to_next_voxel(const vec3& pos, const vec3& dir, const vec3& idir, float res) { // dda like step
	vec3 p = res * (pos - vec3(0.5f));
	float tx = (floorf(p.x + 0.5f + 0.5f * sign(dir.x)) - p.x) * idir.x;
	float ty = (floorf(p.y + 0.5f + 0.5f * sign(dir.y)) - p.y) * idir.y;
	float tz = (floorf(p.z + 0.5f + 0.5f * sign(dir.z)) - p.z) * idir.z;
	float t = min(min(tx, ty), tz);

	return fmaxf(t / res, 0.0f);
}




inline __device__ float advance_to_next_voxel(float t, float cone_angle, const vec3& pos, const vec3& dir, const vec3& idir, uint32_t mip) {
	float res = scalbnf(NERF_GRIDSIZE(), -(int)mip);

	float t_target = t + distance_to_next_voxel(pos, dir, idir, res);

	// Analytic stepping in multiples of 1 in the "log-space" of our exponential stepping routine
	t = to_stepping_space(t, cone_angle);
	t_target = to_stepping_space(t_target, cone_angle);

	return from_stepping_space(t + ceilf(fmaxf(t_target - t, 0.5f)), cone_angle);
}


__device__ inline float network_to_rgb(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ inline float network_to_rgb_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -10.0f, 10.0f));
		default: assert(false);
	}
	return 0.0f;
}

template <typename T>
__device__ inline vec3 network_to_rgb_derivative_vec(const T& val, ENerfActivation activation) {
	return {
		network_to_rgb_derivative(float(val[0]), activation),
		network_to_rgb_derivative(float(val[1]), activation),
		network_to_rgb_derivative(float(val[2]), activation),
	};
}



__device__ inline float network_to_density(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return val;
		case ENerfActivation::ReLU: return val > 0.0f ? val : 0.0f;
		case ENerfActivation::Logistic: return tcnn::logistic(val);
		case ENerfActivation::Exponential: return __expf(val);
		default: assert(false);
	}
	return 0.0f;
}

__device__ inline float network_to_density_derivative(float val, ENerfActivation activation) {
	switch (activation) {
		case ENerfActivation::None: return 1.0f;
		case ENerfActivation::ReLU: return val > 0.0f ? 1.0f : 0.0f;
		case ENerfActivation::Logistic: { float density = tcnn::logistic(val); return density * (1 - density); };
		case ENerfActivation::Exponential: return __expf(tcnn::clamp(val, -15.0f, 15.0f));
		default: assert(false);
	}
	return 0.0f;
}

__device__ inline Array3f network_to_rgb(const tcnn::vector_t<tcnn::network_precision_t, 4>& local_network_output, ENerfActivation activation) {
	return {
		network_to_rgb(float(local_network_output[0]), activation),
		network_to_rgb(float(local_network_output[1]), activation),
		network_to_rgb(float(local_network_output[2]), activation)
	};
}


template <typename T>
__device__ inline vec3 network_to_rgb_vec(const T& val, ENerfActivation activation) {
	return {
		network_to_rgb(float(val[0]), activation),
		network_to_rgb(float(val[1]), activation),
		network_to_rgb(float(val[2]), activation),
	};
}




__device__ inline vec3 warp_position(const vec3& pos, const BoundingBox& aabb) {
	// return {tcnn::logistic(pos.x - 0.5f), tcnn::logistic(pos.y - 0.5f), tcnn::logistic(pos.z - 0.5f)};
	// return pos;
	return aabb.relative_pos(pos);
}


__device__ inline vec3 unwarp_position(const vec3& pos, const BoundingBox& aabb) {
	// return {logit(pos.x) + 0.5f, logit(pos.y) + 0.5f, logit(pos.z) + 0.5f};
	// return pos;

	return aabb.min + pos * aabb.diag();
}

__device__ inline vec3 unwarp_position_derivative(const vec3& pos, const BoundingBox& aabb) {
	// return {logit(pos.x()) + 0.5f, logit(pos.y()) + 0.5f, logit(pos.z()) + 0.5f};
	// return pos;

	return aabb.diag();
}

__device__ inline vec3 warp_position_derivative(const vec3& pos, const BoundingBox& aabb) {
	return vec3(1.0f) / unwarp_position_derivative(pos, aabb);
}

__host__ __device__ inline vec3 warp_direction(const vec3& dir) {
	return (dir + vec3(1.0f)) * 0.5f;
}

__device__ inline vec3 unwarp_direction(const vec3& dir) {
	return dir * 2.0f - vec3(1.0f);
}


__device__ inline vec3 warp_direction_derivative(const vec3& dir) {
	return vec3(0.5f);
}

__device__ inline vec3 unwarp_direction_derivative(const vec3& dir) {
	return vec3(2.0f);
}

__device__ inline float warp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return (dt - MIN_CONE_STEPSIZE()) / (max_stepsize - MIN_CONE_STEPSIZE());
}

__device__ inline float unwarp_dt(float dt) {
	float max_stepsize = MIN_CONE_STEPSIZE() * (1<<(NERF_CASCADES()-1));
	return dt * (max_stepsize - MIN_CONE_STEPSIZE()) + MIN_CONE_STEPSIZE();
}


__device__ inline uint32_t cascaded_grid_idx_at(vec3 pos, uint32_t mip) {
	float mip_scale = scalbnf(1.0f, -mip);
	pos -= vec3(0.5f);
	pos *= mip_scale;
	pos += vec3(0.5f);

	ivec3 i = pos * (float)NERF_GRIDSIZE();
	if (i.x < 0 || i.x >= NERF_GRIDSIZE() || i.y < 0 || i.y >= NERF_GRIDSIZE() || i.z < 0 || i.z >= NERF_GRIDSIZE()) {
		return 0xFFFFFFFF;
	}

	return tcnn::morton3D(i.x, i.y, i.z);
}


__device__ inline bool density_grid_occupied_at(const vec3& pos, const uint8_t* density_grid_bitfield, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	if (idx == 0xFFFFFFFF) {
		return false;
	}
	return density_grid_bitfield[idx/8+grid_mip_offset(mip)/8] & (1<<(idx%8));
}


__device__ inline float cascaded_grid_at(vec3 pos, const float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	if (idx == 0xFFFFFFFF) {
		return 0.0f;
	}
	return cascaded_grid[idx+grid_mip_offset(mip)];
}


__device__ inline float& cascaded_grid_at(vec3 pos, float* cascaded_grid, uint32_t mip) {
	uint32_t idx = cascaded_grid_idx_at(pos, mip);
	if (idx == 0xFFFFFFFF) {
		idx = 0;
		printf("WARNING: invalid cascaded grid access.");
	}
	return cascaded_grid[idx+grid_mip_offset(mip)];
}










inline __device__ int mip_from_pos(const Vector3f& pos, uint32_t max_cascade = NERF_CASCADES()-1) {
	int exponent;
	float maxval = (pos - Vector3f::Constant(0.5f)).cwiseAbs().maxCoeff();
	frexpf(maxval, &exponent);
	return min(max_cascade, max(0, exponent+1));
}

inline __device__ int mip_from_dt(float dt, const Vector3f& pos, uint32_t max_cascade = NERF_CASCADES()-1) {
	int mip = mip_from_pos(pos, max_cascade);
	dt *= 2*NERF_GRIDSIZE();
	if (dt<1.f) return mip;
	int exponent;
	frexpf(dt, &exponent);
	return min(max_cascade, max(exponent, mip));
}


inline __device__ float pdf_2d(Vector2f sample, uint32_t img, const Vector2i& res, const float* __restrict__ cdf_x_cond_y, const float* __restrict__ cdf_y) {
	Vector2i p = (sample.cwiseProduct(res.cast<float>())).cast<int>().cwiseMax(0).cwiseMin(res - Vector2i::Ones());

	cdf_y += img * res.y();
	cdf_x_cond_y += img * res.y() * res.x() + p.y() * res.x();

	float pmf_y = cdf_y[p.y()];
	if (p.y() > 0) {
		pmf_y -= cdf_y[p.y()-1];
	}

	float pmf_x = cdf_x_cond_y[p.x()];
	if (p.x() > 0) {
		pmf_x -= cdf_x_cond_y[p.x()-1];
	}

	// Probability mass of picking the pixel
	float pmf = pmf_x * pmf_y;

	// To convert to probability density, divide by area of pixel
	return UNIFORM_SAMPLING_FRACTION + pmf * res.prod() * (1.0f - UNIFORM_SAMPLING_FRACTION);
}


inline __device__ Vector2f nerf_random_image_pos_for_tracking(default_rng_t& rng, const Vector2i& resolution, bool snap_to_pixel_centers, const Vector2i& margins, const Vector2i& half_kernel_size) {
	Vector2f xy = random_val_2d(rng);

    Vector2i bounds = resolution - 2*margins - 2*half_kernel_size;

    Vector2i xy_int = xy.cwiseProduct(bounds.cast<float>()).cast<int>();
    xy_int = xy_int + margins + half_kernel_size;
    xy = xy_int.cwiseMax(margins + half_kernel_size).cwiseMin(resolution - margins - half_kernel_size - Vector2i::Ones()).cast<float>();

	if (snap_to_pixel_centers) {
		xy = (xy + Vector2f::Constant(0.5f)).cwiseQuotient(resolution.cast<float>());
	} else {
		xy = xy.cwiseQuotient(resolution.cast<float>());
    }
    return xy;
}


inline __device__ uint32_t image_idx(uint32_t base_idx, uint32_t n_rays, uint32_t n_rays_total, uint32_t n_training_images, const float* __restrict__ cdf = nullptr, float* __restrict__ pdf = nullptr) {
	if (cdf) {
		float sample = ld_random_val(base_idx/* + n_rays_total*/, 0xdeadbeef);
		// float sample = random_val(base_idx/* + n_rays_total*/);
		uint32_t img = binary_search(sample, cdf, n_training_images);

		if (pdf) {
			float prev = img > 0 ? cdf[img-1] : 0.0f;
			*pdf = (cdf[img] - prev) * n_training_images;
		}

		return img;
	}

	// return ((base_idx/* + n_rays_total*/) * 56924617 + 96925573) % n_training_images;

	// Neighboring threads in the warp process the same image. Increases locality.
	if (pdf) {
		*pdf = 1.0f;
	}
	return (((base_idx/* + n_rays_total*/) * n_training_images) / n_rays) % n_training_images;
}

__device__ inline LossAndGradient loss_and_gradient(const Vector3f& target, const Vector3f& prediction, ELossType loss_type) {
	switch (loss_type) {
		case ELossType::RelativeL2:  return relative_l2_loss(target, prediction); break;
		case ELossType::L1:          return l1_loss(target, prediction); break;
		case ELossType::Mape:        return mape_loss(target, prediction); break;
		case ELossType::Smape:       return smape_loss(target, prediction); break;
		// Note: we divide the huber loss by a factor of 5 such that its L2 region near zero
		// matches with the L2 loss and error numbers become more comparable. This allows reading
		// off dB numbers of ~converged models and treating them as approximate PSNR to compare
		// with other NeRF methods. Self-normalizing optimizers such as Adam are agnostic to such
		// constant factors; optimization is therefore unaffected.
		case ELossType::Huber:       return huber_loss(target, prediction, 0.1f) / 5.0f; break;
		case ELossType::LogL1:       return log_l1_loss(target, prediction); break;
		default: case ELossType::L2: return l2_loss(target, prediction); break;
	}
}




NGP_NAMESPACE_END
