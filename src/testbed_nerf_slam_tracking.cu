#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/encodings/spherical_harmonics.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>


#ifdef copysign
#undef copysign
#endif

using namespace tcnn;

NGP_NAMESPACE_BEGIN

void Testbed::track(uint32_t batch_size) {
	if (!m_training_data_available || m_camera_path.rendering) {
		m_train = false;
		return;
	}

	if (m_testbed_mode == ETestbedMode::None) {
		throw std::runtime_error{"Cannot train without a mode. -> Mode has to be Nerf for SLAM"};
	}

	set_all_devices_dirty();

	// If we don't have a trainer, as can happen when having loaded training data or changed modes without having
	// explicitly loaded a new neural network.
	if (!m_trainer) {
		reload_network_from_file();
		if (!m_trainer) {
			throw std::runtime_error{"Unable to create a neural network trainer."};
		}
	}
	
    if (!m_dlss) {
		// No immediate redraw necessary
		reset_accumulation(false, false);
	}
	
    // Find leaf optimizer and update its settings
	json* leaf_optimizer_config = &m_network_config["optimizer"];
	while (leaf_optimizer_config->contains("nested")) {
		leaf_optimizer_config = &(*leaf_optimizer_config)["nested"];
	}
	(*leaf_optimizer_config)["optimize_matrix_params"] = m_train_network;
	(*leaf_optimizer_config)["optimize_non_matrix_params"] = m_train_encoding;

    if ((m_train_network!=false) or (m_train_encoding!=false)) {
		throw std::runtime_error{"Tracking only. No grid or MLP updates. You'll have to turn off train_nertwork and train_encoding during tracking."};
    }

	m_optimizer->update_hyperparams(m_network_config["optimizer"]);

	bool get_loss_scalar = true;
	{
		
		train_nerf_slam_tracking(batch_size, get_loss_scalar, m_stream.get());

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

}



NGP_NAMESPACE_END