#include "hip/hip_runtime.h"
#include <neural-graphics-primitives/adam_optimizer.h>
#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/json_binding.h>
#include <neural-graphics-primitives/marching_cubes.h>
#include <neural-graphics-primitives/nerf_loader.h>
#include <neural-graphics-primitives/nerf_network.h>
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/encodings/spherical_harmonics.h>
#include <tiny-cuda-nn/loss.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/network.h>
#include <tiny-cuda-nn/optimizer.h>
#include <tiny-cuda-nn/trainer.h>

#include <filesystem/directory.h>
#include <filesystem/path.h>


#ifdef copysign
#undef copysign
#endif

using namespace tcnn;

NGP_NAMESPACE_BEGIN

void Testbed::map(uint32_t batch_size) {
	if (!m_training_data_available || m_camera_path.rendering) {
		m_train = false;
		return;
	}

	if (m_testbed_mode == ETestbedMode::None) {
		throw std::runtime_error{"Cannot train without a mode."};
	}

	set_all_devices_dirty();

	// If we don't have a trainer, as can happen when having loaded training data or changed modes without having
	// explicitly loaded a new neural network.
	if (!m_trainer) {
		reload_network_from_file();
		if (!m_trainer) {
			throw std::runtime_error{"Unable to create a neural network trainer."};
		}
	}

	if (m_testbed_mode == ETestbedMode::Nerf) {
		if (m_nerf.training.optimize_extra_dims) {
			if (m_nerf.training.dataset.n_extra_learnable_dims == 0) {
				m_nerf.training.dataset.n_extra_learnable_dims = 16;
				reset_network();
			}
		}
	}

	if (!m_dlss) {
		// No immediate redraw necessary
		reset_accumulation(false, false);
	}

	uint32_t n_prep_to_skip = m_testbed_mode == ETestbedMode::Nerf ? tcnn::clamp(m_training_step / 16u, 1u, 16u) : 1u;
	if (m_training_step % n_prep_to_skip == 0) {
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_prep_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count() / n_prep_to_skip);
		}};

		switch (m_testbed_mode) {
			case ETestbedMode::Nerf: training_prep_nerf(batch_size, m_stream.get()); break;
			default: throw std::runtime_error{"Invalid training mode (for SLAM it has to be Nerf)."};
		}

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	// Find leaf optimizer and update its settings
	json* leaf_optimizer_config = &m_network_config["optimizer"];
	while (leaf_optimizer_config->contains("nested")) {
		leaf_optimizer_config = &(*leaf_optimizer_config)["nested"];
	}
	(*leaf_optimizer_config)["optimize_matrix_params"] = m_train_network;
	(*leaf_optimizer_config)["optimize_non_matrix_params"] = m_train_encoding;
	m_optimizer->update_hyperparams(m_network_config["optimizer"]);

	//bool get_loss_scalar = m_training_step % 16 == 0;
	bool get_loss_scalar = true;

	{
		auto start = std::chrono::steady_clock::now();
		ScopeGuard timing_guard{[&]() {
			m_training_ms.update(std::chrono::duration<float, std::milli>(std::chrono::steady_clock::now()-start).count());
		}};

		train_nerf_slam(batch_size, get_loss_scalar, m_stream.get());

		CUDA_CHECK_THROW(hipStreamSynchronize(m_stream.get()));
	}

	if (get_loss_scalar) {
		update_loss_graph();
	}
}


NGP_NAMESPACE_END